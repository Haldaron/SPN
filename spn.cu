
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <math.h>
#include <iostream>

typedef struct  node{
	float	val;
	float	dv;
	float	*ws;
	float	*ps;
	float	*dw;
	float	*dp;
	void 	**chs;
	int		x;
	int		y;
	int		w;
	int		h;
} node;


typedef struct layer{
	node **nds;
	void *prev;
	void *next;
} layer;

__global__ void createNode(  int *d_n,node **map, float *input, node ***tree ,int *l_count){
	
	int pos;
	int n=*d_n;
	int i=n-blockDim.x;
	int j=n-blockDim.y;
	int x=threadIdx.x;
	int y=threadIdx.y;
	int t=i+j;
	node *nd;

	nd=&map[j*n+i][y*blockDim.x+x];


	(*nd).h=i+1;
	(*nd).w=j+1;
	(*nd).x=x;
	(*nd).y=y;	

	if(j!=0||i!=0){
		(*nd).val=0;
	}else{
		(*nd).val=1.0f;
	}

	if(j==n && i==n){
		(*nd).dv=1.0f;
	}else{
		(*nd).dv=0.0f;
	}	

	
	(*nd).ps=(*nd).ws+t;
	(*nd).dw=(*nd).ps+t;
	(*nd).dp=(*nd).dw+t;

	for(int k=0;k<t;k++){
		(*nd).ws[k]=1.0f/(t);
	}

	//Vertical split
	for(int k=1;k<j+1;k++){

		(*nd).chs[2*k-2]=&map[(k-1)*n +i][y*(n-i)+x];
		(*nd).chs[2*k-1]=&map[(j-k)*n + i][(y+k)*(n-i)+x];
		(*nd).ps[k-1]=0;
		(*nd).dp[k-1]=0;
		(*nd).dw[k-1]=0;

	}

	//Horizontal split
	for(int k=1;k<i+1;k++){
		(*nd).chs[2*j+2*k-2]=&map[j*n+k-1][y*(n-k+1)+x];
		(*nd).chs[2*j+2*k-1]=&map[j*n+(i-k)][y*(n-i+k)+x+k];
		(*nd).ps[j+k-1]=0;
		(*nd).dp[j+k-1]=0;
		(*nd).dw[j+k-1]=0;

	}

	pos = atomicAdd(&l_count[i+j],1);

	tree[i+j][pos]=nd;

}




__global__ void input(node ***p_layer, char *input){
	int i=threadIdx.x;
	node **layer=*p_layer;
	node *nd=layer[i];
	
	(*nd).val=input[i];
	return;
}


__global__ void updateVal(node ***p_layer){
	int t;
	int i=threadIdx.x;
	float temp;
	node **layer=*p_layer;
	node *nd=layer[i];
	node **chs= (node**)(*nd).chs;
	t=(*nd).w+(*nd).h-2;
	
	float val=0;
	for(int k=0;k<t;k++){
		temp=(*nd).ws[k]*((*chs[2*k]).val)*((*chs[2*k+1]).val);
		(*nd).ps[k]=temp;
		val+=temp;
	}

	(*nd).val=val;
}




__global__ void backProp(node ***p_layer){
	int t;
	int i=threadIdx.x;
	node **layer=*p_layer;
	node *nd=layer[i];
	node **chs= (node**)(*nd).chs;
	t=(*nd).w+(*nd).h-2;

	//Update Product Node derivatives
	for(int k=0;k<t;k++){
		(*nd).dp[k]+=(*nd).dv*(*nd).ws[k];
		(*nd).dw[k]=(*nd).dv*(*nd).ps[k];
	}

	for(int k=0;k<t;k++){
		(*chs[2*k]).dv+=(*nd).dp[k]*(*chs[2*k+1]).val;
		(*chs[2*k+1]).dv+=(*nd).dp[k]*(*chs[2*k]).val;
	}

}


int main(int argc,char **argv)    
{   
	FILE *fd1, *fd2;

	int n,i,j,i2,j2,r,c,tot;
	int *d_n;
	float 		tAllocate,tBuild,tForward,tBackProp;
	hipEvent_t bAllocate,bBuild,bForward,bBackProp;
	hipEvent_t eAllocate,eBuild,eForward,eBackProp;


	if(argc==2){
		n=atoi(argv[1]);
	}else{
		n=5;
	}

	char labels[10000];
	char imgs[n*n*10000];
	char *d_imgs;
	char *d_labels;

	int matrix_bytes=n*n*sizeof(void*);

    hipError_t error;
	float 	*d_input;
	int 	l_count[2*n-1];
	int 	*d_l_count;
	node 	**d_maps;
	node 	***d_tree;


	fd2=fopen("/media/german/Shared/Workspace/Datasets/Dummy/dummy10x10-10000-imgs","r");
	fd1=fopen("/media/german/Shared/Workspace/Datasets/Dummy/dummy10x10-10000-labels","r");

	fread(labels,sizeof(char),10000,fd1);
	fread(imgs,sizeof(char),10000*n*n,fd2);

	fclose(fd1);
	fclose(fd2);

	error=hipMalloc((void **) &d_imgs, 10000*n*n*sizeof(char));
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMalloc((void **) &d_labels, 10000*sizeof(char));
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMemcpy(d_imgs,imgs,10000*n*n*sizeof(char),hipMemcpyHostToDevice);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipMemcpy(d_labels,labels,10000*sizeof(char),hipMemcpyHostToDevice);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}





	memset(l_count,0,(2*n-1)*sizeof(int));
	error =hipEventCreate(&bAllocate);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventCreate(&bBuild);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&bForward);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&bBackProp);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventCreate(&eAllocate);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventCreate(&eBuild);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&eForward);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&eBackProp);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}





	error=hipMalloc((void **) &d_tree, 2*n*sizeof(void*));
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMalloc((void **) &d_maps, matrix_bytes);
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMalloc((void **) &d_input, n*n*sizeof(float));
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMalloc((void **) &d_n, sizeof(int));
	if (error != hipSuccess){
	    printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipMalloc((void **) &d_l_count, 2*(n)*sizeof(int));
	if (error != hipSuccess){
	    printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipMemset( d_l_count, 0,2*(n)*sizeof(int));
	if (error != hipSuccess){
	    printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipMemcpy(d_n,&n,sizeof(int),hipMemcpyHostToDevice);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}



	error=hipEventRecord(bAllocate,0);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}	

	tot=0;
	for(i=0; i<n;i++){
		for(j=0; j<n;j++){

			r=n-i;
			c=n-j;
			node *mat;

			error=hipMalloc(&mat,r*c*sizeof(node));
			if (error != hipSuccess){
		        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		    }

		    error=hipMemcpy(&d_maps[i*n+j],&mat,sizeof(void*), hipMemcpyHostToDevice);
			if (error != hipSuccess){
		        printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		    }
		    l_count[i+j]+=r*c;

		    for(i2=0;i2<r;i2++){
				for(j2=0;j2<c;j2++){


					float *vals;
					node **chs;
					error=hipMalloc(&vals,4*(i+j)*sizeof(float));
					if (error != hipSuccess){
				        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
				    }

				    error=hipMalloc(&chs,2*(i+j)*sizeof(float));
					if (error != hipSuccess){
				        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
				    }

				    error=hipMemcpy(&(mat[j2*r+i2].chs),&chs,sizeof(void*), hipMemcpyHostToDevice);
					if (error != hipSuccess){
				        printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
				    }
					
				    error=hipMemcpy(&(mat[j2*r+i2].ws),&vals,sizeof(void*), hipMemcpyHostToDevice);
					if (error != hipSuccess){
				        printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
				    }
				}			
			}
		}
	}

	printf("building tree\n");

	for(i=0;i<2*n-1;i++){
		node **layer;
		error=hipMalloc(&layer,l_count[i]*sizeof(void*));
		if (error != hipSuccess){
	        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    }

		
	    error=hipMemcpy(&(d_tree[i]),&layer,sizeof(void*), hipMemcpyHostToDevice);
		if (error != hipSuccess){
	        printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    }
	    printf("Layer:%d, %d\n",i,l_count[i]);
	}

	
	error=hipEventRecord(eAllocate,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	printf("-------------------------------------\n");
	printf("TOTAL=%d, (Exp=%d)\n", tot, n*(n+1)*n*(n+1)/4);
	printf("-------------------------------------\n");
	

	printf("%d\n",n );

	error=hipEventRecord(bBuild,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			int r=n-i;
			int c=n-j;
			dim3 THREAD_DIM (r,c);
			createNode<<<1,THREAD_DIM>>>(d_n,(node**)d_maps, d_input, d_tree, d_l_count);
		}
	}
	error = hipDeviceSynchronize();
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipEventRecord(eBuild,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}


	printf("-------------------------------------\n");
	printf("FORWARD\n");
	printf("-------------------------------------\n");

	error=hipEventRecord(bForward,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	input<<<1,l_count[i]>>>((node***)(d_tree+i),d_imgs);

	for(int i=1;i<2*n-1;i++){
		printf("layer: %i, count: %d\n", i, l_count[i]);
		updateVal<<<1,l_count[i]>>>((node***)(d_tree+i));
	}
	error = hipDeviceSynchronize();
	if (error != hipSuccess){
	    printf("Kernel updateVal returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipEventRecord(eForward,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	printf("-------------------------------------\n");
	printf("BackProp\n");
	printf("-------------------------------------\n");

	error=hipEventRecord(bBackProp,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	for(int i=2*n-1;i>0;i--){
		printf("layer: %i, count: %d\n", i, l_count[i]);
		backProp<<<1,l_count[i]>>>((node***)(d_tree+i));
	}

	error = hipDeviceSynchronize();
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipEventRecord(eBackProp,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}




	error =hipEventSynchronize(eAllocate);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventSynchronize(eBuild);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventSynchronize(eForward);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventSynchronize(eBackProp);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}


	error =hipEventElapsedTime(&tAllocate,bAllocate,eAllocate);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventElapsedTime(&tBuild,bBuild,eBuild);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventElapsedTime(&tForward,bForward,eForward);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventElapsedTime(&tBackProp,bBackProp,eBackProp);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	printf("-------------------------------------\n");
	printf("Programa Finalizado\n");
	printf("-------------------------------------\n");
	printf("Tiempos:\n");
	printf("	Allocation:\t%f\n",tAllocate);
	printf("	Build:\t\t%f\n",tBuild);
	printf("	Forward:\t%f\n",tForward);
	printf("	BackProp:\t%f\n",tBackProp);


}

