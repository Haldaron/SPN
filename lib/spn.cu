
#include "spn.h"


typedef struct layer{
	node **nds;
	void *prev;
	void *next;
} layer;



int main(int argc,char **argv)    
{   
	FILE *fd1, *fd2;

	int n,res;
	spn spn1;
	float 		tAllocate,tBuild,tForward,tBackProp;
	hipEvent_t bAllocate,bBuild,bForward,bBackProp;
	hipEvent_t eAllocate,eBuild,eForward,eBackProp;


	if(argc==2){
		n=atoi(argv[1]);
		res=1;
	}else if(argc==3){
		n=atoi(argv[1]);
		res=atoi(argv[2]);
	}else{
		n=5;
		res=1;
	}

	char labels[10000];
	char imgs[n*n*10000];
	char *d_imgs;
	char *d_labels;

	int matrix_bytes=n*n*sizeof(void*);
	float 	*d_input;
	node 	**d_maps;


	fd2=fopen("/media/german/Shared/Workspace/Datasets/Dummy/dummy10x10-10000-imgs","r");
	fd1=fopen("/media/german/Shared/Workspace/Datasets/Dummy/dummy10x10-10000-labels","r");

	fread(labels,sizeof(char),10000,fd1);
	fread(imgs,sizeof(char),10000*n*n,fd2);

	fclose(fd1);
	fclose(fd2);

	error=hipMalloc((void **) &d_imgs, 10000*n*n*sizeof(char));
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMalloc((void **) &d_labels, 10000*sizeof(char));
	if (error != hipSuccess){
        printf("hipMalloc reurned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	error=hipMemcpy(d_imgs,imgs,10000*n*n*sizeof(char),hipMemcpyHostToDevice);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipMemcpy(d_labels,labels,10000*sizeof(char),hipMemcpyHostToDevice);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}



	error =hipEventCreate(&bAllocate);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventCreate(&bBuild);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&bForward);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&bBackProp);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventCreate(&eAllocate);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventCreate(&eBuild);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&eForward);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventCreate(&eBackProp);
	if (error != hipSuccess){
        printf("hipEventCreate returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	
	error=hipMalloc((void **) &d_input, n*n*sizeof(float));
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}




	/*-----------------------------------------------------------------------------------------------------------------
	ALLOCATE
	-----------------------------------------------------------------------------------------------------------------*/

	error=hipEventRecord(bAllocate,0);
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}	


	error=hipMalloc((void **) &d_maps, matrix_bytes);
	if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
	}

	spn_allocate(&spn1,d_maps,n,res);
	printf("TAMAÑO DE LA SPN: %ld",spn1.size);
	
	error=hipEventRecord(eAllocate,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	/*-----------------------------------------------------------------------------------------------------------------
	BUILD
	-----------------------------------------------------------------------------------------------------------------*/


	printf("-------------------------------------\n");
	printf("TOTAL=%d, (Exp=%d)\n", 0, n*(n+1)*n*(n+1)/4);
	printf("-------------------------------------\n");
	

	printf("%d\n",n );

	error=hipEventRecord(bBuild,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	//spn_build(&spn1, d_maps);
	
	error = hipDeviceSynchronize();
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	error=hipEventRecord(eBuild,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}


	printf("-------------------------------------\n");
	printf("FORWARD\n");
	printf("-------------------------------------\n");

	error=hipEventRecord(bForward,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}
	for(int i=1;i<2*(n/res)+2*res-1;i++){
		updateVal<<<1,(spn1.l_count)[i]>>>((node***)((spn1.tree)+i));
	}
	error = hipDeviceSynchronize();	
	if (error != hipSuccess){
		    printf("Kernel updateVal returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		    exit(EXIT_FAILURE);
	}
		
	


	error=hipEventRecord(eForward,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}

	printf("-------------------------------------\n");
	printf("BackProp\n");
	printf("-------------------------------------\n");

	error=hipEventRecord(bBackProp,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}


	for(int i=2*(n/res)+2*res-4;i>0;i--){
		backProp<<<1,(spn1.l_count)[i]>>>((node***)((spn1.tree)+i));
	
	}
	error = hipDeviceSynchronize();
	if (error != hipSuccess){
	    printf("hipMemcpy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}
	
	error=hipEventRecord(eBackProp,0);
	if (error != hipSuccess){
	    printf("hipEventRecord returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}




	error =hipEventSynchronize(eAllocate);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventSynchronize(eBuild);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	
	error =hipEventSynchronize(eForward);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventSynchronize(eBackProp);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}


	error =hipEventElapsedTime(&tAllocate,bAllocate,eAllocate);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventElapsedTime(&tBuild,bBuild,eBuild);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventElapsedTime(&tForward,bForward,eForward);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}

	error =hipEventElapsedTime(&tBackProp,bBackProp,eBackProp);
	if (error != hipSuccess){
        printf("hipEventSynchronize returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__-2);
        exit(EXIT_FAILURE);
	}
	printf("-------------------------------------\n");
	printf("Programa Finalizado\n");
	printf("-------------------------------------\n");
	printf("Tiempos:\n");
	printf("	Allocation:\t%f\n",tAllocate);
	printf("	Build:\t\t%f\n",tBuild);
	printf("	Forward:\t%f\n",tForward);
	printf("	BackProp:\t%f\n",tBackProp);
	printf("	Total:\t%f\n",tBackProp+tForward);


}

